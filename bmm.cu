#include "hip/hip_runtime.h"
#include "bmm.h"
#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z


#define TILEX 32
#define TILEY 32
// check TILEX and TILEY value for optimal TILE assignment 
// consider that this part is done in preprosecing and in
// compile time. so this assignment doesn't have any overhead.
// these DIV values are optimal values that produce minimum
// time for matrix multiplication
const int DIV = (TILEX == 4 && TILEY == 4) ? TILEX / 4 :
		(TILEX == 4 && TILEY == 8) ? TILEX / 2 :
		(TILEX == 4 && TILEY == 16) ? TILEX :
		(TILEX == 4 && TILEY == 32) ? TILEX :
		(TILEX == 8 && TILEY == 4) ? TILEX / 4 :
		(TILEX == 8 && TILEY == 8) ? TILEX / 2 :
		(TILEX == 8 && TILEY == 16) ? TILEX :
		(TILEX == 8 && TILEY == 32) ? TILEX :
		(TILEX == 16 && TILEY == 4) ? TILEX / 8 :
		(TILEX == 16 && TILEY == 8) ? TILEX / 4 :
		(TILEX == 16 && TILEY == 16) ? TILEX :
		(TILEX == 16 && TILEY == 32) ? TILEX :
		(TILEX == 32 && TILEY == 4) ? TILEX / 8 :
		(TILEX == 32 && TILEY == 8) ? TILEX / 8 :
		(TILEX == 32 && TILEY == 16) ? TILEX / 2 : TILEX / 4;

// with repsect to DIV, assign TILE size
const int T = (TILEX * TILEY) / DIV;

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {
	// shared memory def:
	__shared__ float as[TILEY][T];
	__shared__ float bs[T][TILEX];
	
	// number of read for each of matrices
	int Ra = TILEY / DIV;
	int Rb = TILEX / DIV;
	
	//global index
	int i = ty + by * TILEY;
	int j = tx + bx * TILEX;
		
	float s = 0;
	for(int k = 0; k < n / T; k++){		
		// as read:
		for(int m = 0; m < Ra; m++)
			as[ty][Ra * tx + m] = ad[(i * n) + Ra * tx + k * T + m];		
		// bs read:
		for(int m = 0; m < Rb; m++)
			bs[Rb * ty + m][tx] = bd[(ty * Rb + k * T + m) * n + j];
		__syncthreads();
		// calculation
		for (int m = 0; m < T; m++)
			s += as[ty][m] * bs[m][tx];
		__syncthreads();
	}
	cd[i * n + j] = s;
}

